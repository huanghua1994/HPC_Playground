#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>

#include "cuda_utils.cuh"
#include <hipblas.h>
#include <hiprand.h>
#include <hip/hip_fp16.h>

template <typename T>
void test_gemm(const int m, const int n, const int k, const int n_test)
{
    const T T_zero = 0, T_one = 1;
    const int ldA = m, ldB = k, ldC = m;

    hipStream_t stream;
    hipblasHandle_t cublas_handle;
    CUDA_CHECK( hipStreamCreate(&stream) );
    CUBLAS_CHECK( hipblasCreate(&cublas_handle) );
    CUBLAS_CHECK( hipblasSetStream(cublas_handle, stream) );

    T *A = nullptr, *B = nullptr, *C = nullptr;
    CUDA_CHECK( hipMalloc((void **) &A, sizeof(T) * m * k) );
    CUDA_CHECK( hipMalloc((void **) &B, sizeof(T) * k * n) );
    CUDA_CHECK( hipMalloc((void **) &C, sizeof(T) * m * n) );

    constexpr hipDataType dtype_T = T_to_cuda_dtype<T>();

    hiprandGenerator_t gen;
    CURAND_CHECK( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CHECK( hiprandSetPseudoRandomGeneratorSeed(gen, 19241112) );
    CURAND_CHECK( hiprandGenerate(gen, (unsigned int *) A, sizeof(T) * m * k / sizeof(unsigned int)) );
    CURAND_CHECK( hiprandGenerate(gen, (unsigned int *) B, sizeof(T) * k * n / sizeof(unsigned int)) );
    CURAND_CHECK( hiprandDestroyGenerator(gen) );
    CUDA_CHECK( hipMemset(C, 0, sizeof(T) * m * n) );

    hipEvent_t gemm_start, gemm_stop;
    CUDA_CHECK( hipEventCreate(&gemm_start) );
    CUDA_CHECK( hipEventCreate(&gemm_stop) );
    fprintf(stderr, "gemm_ms, gemm_gflops\n");
    for (int i_test = 0; i_test < n_test; i_test++)
    {
        CUDA_CHECK( hipEventRecord(gemm_start, stream) );
        CUBLAS_CHECK( hipblasGemmEx(
            cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
            &T_one, A, dtype_T, ldA, B, dtype_T, ldB,
            &T_zero, C, dtype_T, ldC, dtype_T, HIPBLAS_GEMM_DEFAULT
        ) );
        CUDA_CHECK( hipEventRecord(gemm_stop, stream) );
        CUDA_CHECK( hipEventSynchronize(gemm_start) );
        CUDA_CHECK( hipEventSynchronize(gemm_stop) );

        float gemm_ms;
        float mflops = (2.0f * (float) m * (float) n * (float) k) / 1000000.0f;
        CUDA_CHECK( hipEventElapsedTime(&gemm_ms, gemm_start, gemm_stop) );
        fprintf(stderr, "%.2f, %.2f\n", gemm_ms, mflops / gemm_ms);
    }

    CUDA_CHECK( hipEventDestroy(gemm_start) );
    CUDA_CHECK( hipEventDestroy(gemm_stop) );

    CUDA_CHECK( hipFree(A) );
    CUDA_CHECK( hipFree(B) );
    CUDA_CHECK( hipFree(C) );

    CUDA_CHECK( hipStreamDestroy(stream) );
    CUBLAS_CHECK( hipblasDestroy(cublas_handle) );
}

int main(int argc, char **argv)
{
    int dtype = 0, m = 0, n = 0, k = 0, n_test = 10;
    if (argc < 5)
    {
        fprintf(stderr, "Usage: %s dtype m n k n_test \n", argv[0]);
        fprintf(stderr, "  - dtype    : 0 for double, 1 for float, 2 for half\n");
        fprintf(stderr, "  - m, n, k  : Matrix A: m * k, matrix B: k * n, matrix C: m * n\n");
        fprintf(stderr, "  - n_test   : Number of tests to run\n");
        return 255;
    }
    dtype = atoi(argv[1]);
    if (dtype < 0 || dtype > 2) dtype = 0;
    m = atoi(argv[2]);
    n = atoi(argv[3]);
    k = atoi(argv[4]);
    if (argc >= 6) n_test = atoi(argv[5]);
    fprintf(
        stderr, "dtype = %d, m = %d, n = %d, k = %d, n_test = %d\n", 
        dtype, m, n, k, n_test
    );

    if (dtype == 0) test_gemm<double>(m, n, k, n_test);
    if (dtype == 1) test_gemm<float> (m, n, k, n_test);
    if (dtype == 2) test_gemm<__half>(m, n, k, n_test);

    return 0;
}