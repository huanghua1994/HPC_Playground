#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cuda_utils.cuh"
#include "cublas_cusolver_init.cuh"
#include <hiprand.h>

void test_cublas_gemm(const int m, const int n, const int k, const int n_test)
{
    init_cublas_cusolver();

    int ldA = m, ldB = k, ldC = m;
    double *A = NULL, *B = NULL, *C = NULL;
    CUDA_RT_CHECK( hipMalloc((void **) &A, sizeof(double) * m * k) );
    CUDA_RT_CHECK( hipMalloc((void **) &B, sizeof(double) * k * n) );
    CUDA_RT_CHECK( hipMalloc((void **) &C, sizeof(double) * m * n) );
 
    hiprandGenerator_t gen;
    CURAND_CHECK( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CHECK( hiprandSetPseudoRandomGeneratorSeed(gen, 19241112) );
    CURAND_CHECK( hiprandGenerateUniformDouble(gen, A, m * k) );
    CURAND_CHECK( hiprandGenerateUniformDouble(gen, B, k * n) );
    CUDA_RT_CHECK( hipMemset(C, 0, sizeof(double) * m * n) );
    CURAND_CHECK( hiprandDestroyGenerator(gen) );

    hipEvent_t gemm_start, gemm_stop;
    CUDA_RT_CHECK( hipEventCreate(&gemm_start) );
    CUDA_RT_CHECK( hipEventCreate(&gemm_stop) );
    fprintf(stderr, "gemm_ms, gemm_gflops\n");
    for (int i_test = 0; i_test < n_test; i_test++)
    {
        CUDA_RT_CHECK( hipEventRecord(gemm_start, cublas_stream) );
        CUBLAS_CHECK( hipblasDgemm(
            cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, 
            &d_one, A, ldA, B, ldB, &d_one, C, ldC
        ) );
        CUDA_RT_CHECK( hipEventRecord(gemm_stop, cublas_stream) );
        CUDA_RT_CHECK( hipEventSynchronize(gemm_start) );
        CUDA_RT_CHECK( hipEventSynchronize(gemm_stop) );

        float gemm_ms;
        float mflops = (2.0f * (float) m * (float) n * (float) k) / 1000000.0f;
        CUDA_RT_CHECK( hipEventElapsedTime(&gemm_ms, gemm_start, gemm_stop) );
        fprintf(stderr, "%.2f, %.2f\n", gemm_ms, mflops / gemm_ms);
    }

    CUDA_RT_CHECK( hipFree(A) );
    CUDA_RT_CHECK( hipFree(B) );
    CUDA_RT_CHECK( hipFree(C) );
    CUDA_RT_CHECK( hipEventDestroy(gemm_start) );
    CUDA_RT_CHECK( hipEventDestroy(gemm_stop) );
}

int main(int argc, char **argv)
{
    int m = 0, n = 0, k = 0, n_test = 10;
    if (argc < 5)
    {
        fprintf(stderr, "Usage: %s m n k n_test \n", argv[0]);
        return 255;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    k = atoi(argv[3]);
    n_test = atoi(argv[4]);
    fprintf(stderr, "m = %d, n = %d, k = %d, n_test = %d\n", m, n, k, n_test);

    test_cublas_gemm(m, n, k, n_test);

    return 0;
}