#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cuda_utils.cuh"
#include "cublas_cusolver_init.cuh"
#include <hiprand.h>

void test_cusolver_qr(const int m, const int n, const int n_test)
{
    init_cublas_cusolver();

    int ldA = m;
    int lwork_geqrf = 0, lwork_orgqr = 0, retval;
    double *A = NULL, *A0 = NULL, *Q = NULL, *tau = NULL;
    double *geqrf_work = NULL, *orgqr_work = NULL;

    // Allocate memory on device
    CUDA_RT_CHECK( hipMalloc((void **) &A,   sizeof(double) * m * n) );
    CUDA_RT_CHECK( hipMalloc((void **) &A0,  sizeof(double) * m * n) );
    CUDA_RT_CHECK( hipMalloc((void **) &Q,   sizeof(double) * m * n) );
    CUDA_RT_CHECK( hipMalloc((void **) &tau, sizeof(double) * n) );
    CUSOLVER_CHECK( hipsolverDnDgeqrf_bufferSize(cusolver_dn_handle, m, n, A, ldA, &lwork_geqrf) );
    CUSOLVER_CHECK( hipsolverDnDorgqr_bufferSize(cusolver_dn_handle, m, n, n, Q, ldA, tau, &lwork_orgqr) );
    CUDA_RT_CHECK( hipMalloc((void **) &geqrf_work, sizeof(double) * lwork_geqrf) );
    CUDA_RT_CHECK( hipMalloc((void **) &orgqr_work, sizeof(double) * lwork_orgqr) );
 
    // Initialize random A
    hiprandGenerator_t gen;
    CURAND_CHECK( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CHECK( hiprandSetPseudoRandomGeneratorSeed(gen, 19241112) );
    CURAND_CHECK( hiprandGenerateUniformDouble(gen, A0, m * n) );
    CURAND_CHECK( hiprandDestroyGenerator(gen) );

    // Test dgeqrf and dorgqr
    hipEvent_t dgeqrf_start, dgeqrf_stop, dorgqr_start, dorgqr_stop;
    CUDA_RT_CHECK( hipEventCreate(&dgeqrf_start) );
    CUDA_RT_CHECK( hipEventCreate(&dgeqrf_stop) );
    CUDA_RT_CHECK( hipEventCreate(&dorgqr_start) );
    CUDA_RT_CHECK( hipEventCreate(&dorgqr_stop) );
    fprintf(stderr, "geqrf_ms, orgqr_ms | geqrf_gflops, orgqr_gflops\n");
    for (int i_test = 0; i_test < n_test; i_test++)
    {
        CUDA_RT_CHECK( hipMemcpyAsync(A, A0, sizeof(double) * m * n, hipMemcpyDeviceToDevice, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(dgeqrf_start, cusolver_stream) );
        CUSOLVER_CHECK( hipsolverDnDgeqrf(
            cusolver_dn_handle, m, n, A, ldA, tau,
            geqrf_work, lwork_geqrf, cusolver_retval
        ) );
        CUDA_RT_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(dgeqrf_stop, cusolver_stream) );
        CUDA_RT_CHECK( hipEventSynchronize(dgeqrf_start) );
        CUDA_RT_CHECK( hipEventSynchronize(dgeqrf_stop) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDgeqrf returned %d\n", __FILE__, __LINE__, retval);
        
        CUDA_RT_CHECK( hipMemcpyAsync(Q, A, sizeof(double) * m * n, hipMemcpyDeviceToDevice, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(dorgqr_start, cusolver_stream) );
        CUSOLVER_CHECK( hipsolverDnDorgqr(
            cusolver_dn_handle, m, n, n, Q, ldA, tau,
            orgqr_work, lwork_orgqr, cusolver_retval
        ) );
        CUDA_RT_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(dorgqr_stop, cusolver_stream) );
        CUDA_RT_CHECK( hipEventSynchronize(dorgqr_start) );
        CUDA_RT_CHECK( hipEventSynchronize(dorgqr_stop) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDorgqr returned %d\n", __FILE__, __LINE__, retval);

        float geqrf_ms, orgqr_ms;
        float mflops = (2.0f * (float) m * (float) n * (float) n - (2.0f/3.0f) * (float) n * (float) n * (float) n) / 1000000.0f;
        CUDA_RT_CHECK( hipEventElapsedTime(&geqrf_ms, dgeqrf_start, dgeqrf_stop) );
        CUDA_RT_CHECK( hipEventElapsedTime(&orgqr_ms, dorgqr_start, dorgqr_stop) );
        fprintf(stderr, "%.2f, %.2f | %.2f, %.2f\n", geqrf_ms, orgqr_ms, mflops / geqrf_ms, mflops / orgqr_ms);
    }

    CUDA_RT_CHECK( hipFree(A) );
    CUDA_RT_CHECK( hipFree(A0) );
    CUDA_RT_CHECK( hipFree(Q) );
    CUDA_RT_CHECK( hipFree(tau) );
    CUDA_RT_CHECK( hipFree(geqrf_work) );
    CUDA_RT_CHECK( hipFree(orgqr_work) );
    CUDA_RT_CHECK( hipEventDestroy(dgeqrf_start) );
    CUDA_RT_CHECK( hipEventDestroy(dgeqrf_stop) );
    CUDA_RT_CHECK( hipEventDestroy(dorgqr_start) );
    CUDA_RT_CHECK( hipEventDestroy(dorgqr_stop) );
}

int main(int argc, char **argv)
{
    int m = 0, n = 0, n_test = 10;
    if (argc < 4)
    {
        fprintf(stderr, "Usage: %s m n n_test \n", argv[0]);
        return 255;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    n_test = atoi(argv[3]);
    fprintf(stderr, "m = %d, n = %d, n_test = %d\n", m, n, n_test);

    test_cusolver_qr(m, n, n_test);

    return 0;
}