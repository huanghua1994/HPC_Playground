#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cuda_utils.cuh"
#include "cublas_cusolver_init.cuh"
#include <hiprand.h>

void test_cusolver_syevd(const int m, const int n_test)
{
    init_cublas_cusolver();

    int ldA = m;
    int lwork_syevd = 0, retval;
    double *A = NULL, *W = NULL, *A0 = NULL, *syevd_work = NULL;

    // Allocate memory on device
    CUDA_CHECK( hipMalloc((void **) &A,  sizeof(double) * m * m) );
    CUDA_CHECK( hipMalloc((void **) &A0, sizeof(double) * m * m) );
    CUDA_CHECK( hipMalloc((void **) &W,  sizeof(double) * m) );
    CUSOLVER_CHECK( hipsolverDnDsyevd_bufferSize(
        cusolver_dn_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 
        m, A, ldA, W, &lwork_syevd
    ) );
    CUDA_CHECK( hipMalloc((void **) &syevd_work, sizeof(double) * lwork_syevd) );

    // Initialize random SPD A
    hiprandGenerator_t gen;
    CURAND_CHECK( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CHECK( hiprandSetPseudoRandomGeneratorSeed(gen, 19241112) );
    CURAND_CHECK( hiprandGenerateUniformDouble(gen, A, m * m) );
    CURAND_CHECK( hiprandDestroyGenerator(gen) );
    CUBLAS_CHECK( hipblasDgemm(
        cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, m, m, 
        &d_one, A, ldA, A, ldA, &d_zero, A0, ldA
    ) );

    // Test dgeqrf and dorgqr
    hipEvent_t syevd_start, syevd_stop;
    CUDA_CHECK( hipEventCreate(&syevd_start) );
    CUDA_CHECK( hipEventCreate(&syevd_stop) );
    fprintf(stderr, "syevd_ms\n");
    for (int i_test = 0; i_test < n_test; i_test++)
    {
        CUDA_CHECK( hipMemcpyAsync(A, A0, sizeof(double) * m * m, hipMemcpyDeviceToDevice, cusolver_stream) );
        CUDA_CHECK( hipEventRecord(syevd_start, cusolver_stream) );
        CUSOLVER_CHECK( hipsolverDnDsyevd(
            cusolver_dn_handle, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 
            m, A, ldA, W, syevd_work, lwork_syevd, cusolver_retval
        ) );
        CUDA_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_CHECK( hipEventRecord(syevd_stop, cusolver_stream) );
        CUDA_CHECK( hipEventSynchronize(syevd_start) );
        CUDA_CHECK( hipEventSynchronize(syevd_stop) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDsyevd returned %d\n", __FILE__, __LINE__, retval);

        float syevd_ms;
        CUDA_CHECK( hipEventElapsedTime(&syevd_ms, syevd_start, syevd_stop) );
        fprintf(stderr, "%.2f\n", syevd_ms);
    }

    CUDA_CHECK( hipFree(A) );
    CUDA_CHECK( hipFree(A0) );
    CUDA_CHECK( hipFree(W) );
    CUDA_CHECK( hipFree(syevd_work) );
    CUDA_CHECK( hipEventDestroy(syevd_start) );
    CUDA_CHECK( hipEventDestroy(syevd_stop) );
}

int main(int argc, char **argv)
{
    int m = 0, n_test = 10;
    if (argc < 3)
    {
        fprintf(stderr, "Usage: %s m n_test \n", argv[0]);
        return 255;
    }
    m = atoi(argv[1]);
    n_test = atoi(argv[2]);
    fprintf(stderr, "m = %d, n_test = %d\n", m, n_test);

    test_cusolver_syevd(m, n_test);

    return 0;
}