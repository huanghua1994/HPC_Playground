#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cuda_utils.cuh"
#include "cublas_cusolver_init.cuh"
#include <hiprand.h>

__global__ void set_diag_element_kernel(const int ncol, double *A, const int ldA, const double val)
{
    for (int i = threadIdx.x; i < ncol; i += blockDim.x) A[i * ldA + i] = val;
    __syncthreads();
}

void test_cusolver_lu_chol(const int m, const int n_test)
{
    init_cublas_cusolver();

    int ldA = m;
    int lwork_getrf = 0, lwork_potrf = 0, lwork_potri = 0, retval;
    double *A = NULL, *A0 = NULL, *B = NULL;
    double *getrf_work = NULL, *potrf_work = NULL, *potri_work = NULL;
    int *ipiv;

    // Allocate memory on device
    CUDA_RT_CHECK( hipMalloc((void **) &A,  sizeof(double) * m * m) );
    CUDA_RT_CHECK( hipMalloc((void **) &A0, sizeof(double) * m * m) );
    CUDA_RT_CHECK( hipMalloc((void **) &B,  sizeof(double) * m * m) );
    CUDA_RT_CHECK( hipMalloc((void **) &ipiv, sizeof(int) * m) );
    CUSOLVER_CHECK( hipsolverDnDgetrf_bufferSize(cusolver_dn_handle, m, m, A, ldA, &lwork_getrf) );
    CUSOLVER_CHECK( hipsolverDnDpotrf_bufferSize(cusolver_dn_handle, HIPBLAS_FILL_MODE_UPPER, m, A, ldA, &lwork_potrf) );
    CUSOLVER_CHECK( hipsolverDnDpotri_bufferSize(cusolver_dn_handle, HIPBLAS_FILL_MODE_UPPER, m, A, ldA, &lwork_potri) );
    CUDA_RT_CHECK( hipMalloc((void **) &getrf_work, sizeof(double) * lwork_getrf) );
    CUDA_RT_CHECK( hipMalloc((void **) &potrf_work, sizeof(double) * lwork_potrf) );
    CUDA_RT_CHECK( hipMalloc((void **) &potri_work, sizeof(double) * lwork_potri) );
 
    // Initialize random SPD A
    hiprandGenerator_t gen;
    CURAND_CHECK( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    CURAND_CHECK( hiprandSetPseudoRandomGeneratorSeed(gen, 19241112) );
    CURAND_CHECK( hiprandGenerateUniformDouble(gen, A, m * m) );
    CURAND_CHECK( hiprandDestroyGenerator(gen) );
    CUBLAS_CHECK( hipblasDgemm(
        cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, m, m, 
        &d_one, A, ldA, A, ldA, &d_zero, A0, ldA
    ) );

    // Test dgeqrf and dorgqr
    hipEvent_t *events = (hipEvent_t *) malloc(sizeof(hipEvent_t) * 8);
    for (int i = 0; i < 8; i++) CUDA_RT_CHECK( hipEventCreate(&events[i]) );
    fprintf(stderr, "getrf_ms, getrs_ms, potrf_ms, potri_ms | getrf_gflops, getrs_gflops, potrf_gflops, potri_gflops\n");
    for (int i_test = 0; i_test < n_test; i_test++)
    {
        // LU factorization
        CUDA_RT_CHECK( hipMemcpyAsync(A, A0, sizeof(double) * m * m, hipMemcpyDeviceToDevice, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(events[0], cusolver_stream) );
        CUSOLVER_CHECK( hipsolverDnDgetrf(
            cusolver_dn_handle, m, m, A, ldA, 
            getrf_work, ipiv, cusolver_retval
        ) );
        CUDA_RT_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(events[1], cusolver_stream) );
        CUDA_RT_CHECK( hipEventSynchronize(events[0]) );
        CUDA_RT_CHECK( hipEventSynchronize(events[1]) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDgetrf returned %d\n", __FILE__, __LINE__, retval);

        // Inverse matrix using LU factorization
        CUDA_RT_CHECK( hipEventRecord(events[2], cusolver_stream) );
        CUDA_RT_CHECK( hipMemset(B, 0, sizeof(double) * m * m) );
        set_diag_element_kernel<<<1, 1024>>>(m, B, m, 1.0);
        CUDA_RT_CHECK( hipPeekAtLastError() );
        CUSOLVER_CHECK( hipsolverDnDgetrs(
            cusolver_dn_handle, HIPBLAS_OP_N, m, m, 
            A, ldA, ipiv, B, m, cusolver_retval
        ) );
        CUDA_RT_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(events[3], cusolver_stream) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDgetrs returned %d\n", __FILE__, __LINE__, retval);
        
        // Cholesky factorization
        CUDA_RT_CHECK( hipMemcpyAsync(A, A0, sizeof(double) * m * m, hipMemcpyDeviceToDevice, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(events[4], cusolver_stream) );
        CUSOLVER_CHECK( hipsolverDnDpotrf(
            cusolver_dn_handle, HIPBLAS_FILL_MODE_UPPER, m, A, ldA, 
            potrf_work, lwork_potrf, cusolver_retval
        ) );
        CUDA_RT_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(events[5], cusolver_stream) );
        CUDA_RT_CHECK( hipEventSynchronize(events[4]) );
        CUDA_RT_CHECK( hipEventSynchronize(events[5]) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDpotrf returned %d\n", __FILE__, __LINE__, retval);

        // Inverse matrix using Cholesky factorization
        CUDA_RT_CHECK( hipEventRecord(events[6], cusolver_stream) );
        CUSOLVER_CHECK( hipsolverDnDpotri(
            cusolver_dn_handle, HIPBLAS_FILL_MODE_UPPER, m, A, ldA, 
            potri_work, lwork_potri, cusolver_retval
        ) );
        CUDA_RT_CHECK( hipMemcpyAsync(&retval, cusolver_retval, sizeof(int), hipMemcpyDeviceToHost, cusolver_stream) );
        CUDA_RT_CHECK( hipEventRecord(events[7], cusolver_stream) );
        CUDA_RT_CHECK( hipEventSynchronize(events[6]) );
        CUDA_RT_CHECK( hipEventSynchronize(events[7]) );
        if (retval != 0)
            fprintf(stderr, "[ERROR] %s, %d: hipsolverDnDpotri returned %d\n", __FILE__, __LINE__, retval);

        float getrf_ms, getrs_ms, potrf_ms, potri_ms;
        float getrf_mflops = ((2.0f/3.0f) * (float) m * (float) m * (float) m) / 1000000.0f;
        float getrs_mflops = ((2.0f/1.0f) * (float) m * (float) m * (float) m) / 1000000.0f;
        float potrf_mflops = ((1.0f/3.0f) * (float) m * (float) m * (float) m) / 1000000.0f;
        float potri_mflops = ((2.0f/3.0f) * (float) m * (float) m * (float) m) / 1000000.0f;
        CUDA_RT_CHECK( hipEventElapsedTime(&getrf_ms, events[0], events[1]) );
        CUDA_RT_CHECK( hipEventElapsedTime(&getrs_ms, events[2], events[3]) );
        CUDA_RT_CHECK( hipEventElapsedTime(&potrf_ms, events[4], events[5]) );
        CUDA_RT_CHECK( hipEventElapsedTime(&potri_ms, events[6], events[7]) );
        fprintf(
            stderr, "%.2f, %.2f, %.2f, %.2f | %.2f, %.2f, %.2f, %.2f\n", 
            getrf_ms, getrs_ms, potrf_ms, potri_ms,
            getrf_mflops / getrf_ms, getrs_mflops / getrs_ms,
            potrf_mflops / potrf_ms, potri_mflops / potri_ms
        );
    }

    CUDA_RT_CHECK( hipFree(A) );
    CUDA_RT_CHECK( hipFree(A0) );
    CUDA_RT_CHECK( hipFree(B) );
    CUDA_RT_CHECK( hipFree(ipiv) );
    CUDA_RT_CHECK( hipFree(getrf_work) );
    CUDA_RT_CHECK( hipFree(potrf_work) );
    CUDA_RT_CHECK( hipFree(potri_work) );
    for (int i = 0; i < 8; i++) CUDA_RT_CHECK( hipEventDestroy(events[i]) );
    free(events);
}

int main(int argc, char **argv)
{
    int m = 0, n_test = 10;
    if (argc < 3)
    {
        fprintf(stderr, "Usage: %s m n_test \n", argv[0]);
        return 255;
    }
    m = atoi(argv[1]);
    n_test = atoi(argv[2]);
    fprintf(stderr, "m = %d, n_test = %d\n", m, n_test);

    test_cusolver_lu_chol(m, n_test);

    return 0;
}