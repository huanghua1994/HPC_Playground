#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include "CUDA_Utils.h"

extern "C" {
#include "cuSPARSE_SpMV_test.h"
}

extern "C"
void cuSPARSE_SpMV_test(
    const int nrows, const int ncols, const int nnz, 
    const int *row_ptr, const int *col, const double *val, 
    const double *x, double *y, const int ntest
)
{
    int nrows1 = nrows + 1;
    const double alpha = 1.0;
    const double beta  = 0.0;
    
    // Allocate memory on device
    int *cu_row_ptr, *cu_col;
    double *cu_val, *cu_x, *cu_y;
    cudaCheck( hipMalloc(&cu_row_ptr, sizeof(int)    * nrows1) );
    cudaCheck( hipMalloc(&cu_col,     sizeof(int)    * nnz)    );
    cudaCheck( hipMalloc(&cu_val,     sizeof(double) * nnz)    );
    cudaCheck( hipMalloc(&cu_x,       sizeof(double) * ncols)  );
    cudaCheck( hipMalloc(&cu_y,       sizeof(double) * nrows)  );
    
    // Copy CSR matrix to device
    cudaCheck( hipMemcpy(cu_row_ptr, row_ptr, sizeof(int)    * nrows1,    hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_col,     col,     sizeof(int)    * nnz,       hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_val,     val,     sizeof(double) * nnz,       hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_x,       x,       sizeof(double) * ncols,     hipMemcpyHostToDevice) );
    
    // Initialize cuSPARSE
    hipsparseHandle_t   cu_SpMV_handle = 0;
    hipStream_t       cu_SpMV_stream = 0;
    hipsparseMatDescr_t cu_SpMV_descr  = 0;
    cudaSparseCheck( hipsparseCreate(&cu_SpMV_handle)                                  );
    cudaSparseCheck( hipsparseCreateMatDescr(&cu_SpMV_descr)                           );
    cudaSparseCheck( hipsparseSetStream(cu_SpMV_handle, cu_SpMV_stream)                );
    cudaSparseCheck( hipsparseSetMatType(cu_SpMV_descr, HIPSPARSE_MATRIX_TYPE_GENERAL)  );
    cudaSparseCheck( hipsparseSetMatIndexBase(cu_SpMV_descr, HIPSPARSE_INDEX_BASE_ZERO) );
    
    // Run SpMV on GPU
    hipEvent_t st, et;
    float SpMV_ms;
    hipEventCreate(&st);
    hipEventCreate(&et);
    hipEventRecord(st, cu_SpMV_stream);
    for (int i = 0; i < ntest; i++)
    {
        cudaSparseCheck(hipsparseDcsrmv(
            cu_SpMV_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            nrows, ncols, nnz, &alpha, cu_SpMV_descr, cu_val, 
            cu_row_ptr, cu_col, cu_x, &beta, cu_y
        ));
    }
    hipEventRecord(et, cu_SpMV_stream);
    hipEventSynchronize(et);
    hipEventElapsedTime(&SpMV_ms, st, et);
    
    double GFlops, ut;
    GFlops = 2.0 * (double) nnz / 1000000000.0;
    ut = (double) SpMV_ms / (double) (ntest * 1000);
    printf("cuSPARSE  GPU CSR SpMV done, used time = %lf (ms), %lf GFlops\n", ut * 1000.0, GFlops / ut);
    
    // Copy results to host
    cudaCheck( hipMemcpy(y, cu_y, sizeof(double) * nrows, hipMemcpyDeviceToHost) );
    
    // Free CUDA resources
    cudaCheck( hipFree(cu_row_ptr) );
    cudaCheck( hipFree(cu_col)     );
    cudaCheck( hipFree(cu_val)     );
    cudaCheck( hipFree(cu_x)       );
    cudaCheck( hipFree(cu_y)       );
    cudaSparseCheck( hipsparseDestroy(cu_SpMV_handle)        );
    cudaSparseCheck( hipsparseDestroyMatDescr(cu_SpMV_descr) );
}

void cuSPARSE_SpMM_test(
    const int nrows, const int ncols, const int nnz, const int X_ncol, 
    const int *row_ptr, const int *col, const double *val, 
    const double *X, const int ldX, double *Y, const int ldY, const int ntest
)
{
    int nrows1 = nrows + 1;
    const double alpha = 1.0;
    const double beta  = 0.0;
    
    // Allocate memory on device
    int *cu_row_ptr, *cu_col;
    double *cu_val, *cu_X, *cu_Y;
    cudaCheck( hipMalloc(&cu_row_ptr, sizeof(int)    * nrows1) );
    cudaCheck( hipMalloc(&cu_col,     sizeof(int)    * nnz)    );
    cudaCheck( hipMalloc(&cu_val,     sizeof(double) * nnz)    );
    cudaCheck( hipMalloc(&cu_X,       sizeof(double) * ncols * X_ncol)  );
    cudaCheck( hipMalloc(&cu_Y,       sizeof(double) * nrows * X_ncol)  );
    
    // Copy CSR matrix to device
    cudaCheck( hipMemcpy(cu_row_ptr, row_ptr, sizeof(int)    * nrows1,         hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_col,     col,     sizeof(int)    * nnz,            hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_val,     val,     sizeof(double) * nnz,            hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_X,       X,       sizeof(double) * ncols * X_ncol, hipMemcpyHostToDevice) );
    
    // Initialize cuSPARSE
    hipsparseHandle_t   cu_SpMM_handle = 0;
    hipStream_t       cu_SpMM_stream = 0;
    hipsparseMatDescr_t cu_SpMM_descr  = 0;
    cudaSparseCheck( hipsparseCreate(&cu_SpMM_handle)                                  );
    cudaSparseCheck( hipsparseCreateMatDescr(&cu_SpMM_descr)                           );
    cudaSparseCheck( hipsparseSetStream(cu_SpMM_handle, cu_SpMM_stream)                );
    cudaSparseCheck( hipsparseSetMatType(cu_SpMM_descr, HIPSPARSE_MATRIX_TYPE_GENERAL)  );
    cudaSparseCheck( hipsparseSetMatIndexBase(cu_SpMM_descr, HIPSPARSE_INDEX_BASE_ZERO) );
    
    // Run SpMV on GPU
    hipEvent_t st, et;
    float SpMM_ms;
    hipEventCreate(&st);
    hipEventCreate(&et);
    hipEventRecord(st, cu_SpMM_stream);
    for (int i = 0; i < ntest; i++)
    {
        cudaSparseCheck(hipsparseDcsrmm(
            cu_SpMM_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            nrows, X_ncol, ncols, nnz, &alpha, cu_SpMM_descr, 
            cu_val, cu_row_ptr, cu_col, cu_X, ncols, &beta, cu_Y, nrows
        ));
    }
    hipEventRecord(et, cu_SpMM_stream);
    hipEventSynchronize(et);
    hipEventElapsedTime(&SpMM_ms, st, et);
    
    double GFlops, ut;
    GFlops = 2.0 * (double) nnz * (double) X_ncol / 1000000000.0;
    ut = (double) SpMM_ms / (double) (ntest * 1000);
    printf("cuSPARSE  GPU CSR SpMM done, used time = %lf (ms), %lf GFlops\n", ut * 1000.0, GFlops / ut);
    
    // Copy results to host
    cudaCheck( hipMemcpy(Y, cu_Y, sizeof(double) * nrows * X_ncol, hipMemcpyDeviceToHost) );
    
    // Free CUDA resources
    cudaCheck( hipFree(cu_row_ptr) );
    cudaCheck( hipFree(cu_col)     );
    cudaCheck( hipFree(cu_val)     );
    cudaCheck( hipFree(cu_X)       );
    cudaCheck( hipFree(cu_Y)       );
    cudaSparseCheck( hipsparseDestroy(cu_SpMM_handle)        );
    cudaSparseCheck( hipsparseDestroyMatDescr(cu_SpMM_descr) );
}
