#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include "CUDA_Utils.h"

extern "C" {
#include "cuSPARSE_SpMV_test.h"
}

extern "C"
void cuSPARSE_SpMV_test(
    const int nrows, const int ncols, const int nnz, 
    const int *row_ptr, const int *col, const double *val, 
    const double *x, double *y, const int ntest
)
{
    int nrows1 = nrows + 1;
    const double alpha = 1.0;
    const double beta  = 0.0;
    
    // Allocate memory on device
    int *cu_row_ptr, *cu_col;
    double *cu_val, *cu_x, *cu_y;
    cudaCheck( hipMalloc(&cu_row_ptr, sizeof(int)    * nrows1) );
    cudaCheck( hipMalloc(&cu_col,     sizeof(int)    * nnz)    );
    cudaCheck( hipMalloc(&cu_val,     sizeof(double) * nnz)    );
    cudaCheck( hipMalloc(&cu_x,       sizeof(double) * ncols)  );
    cudaCheck( hipMalloc(&cu_y,       sizeof(double) * nrows)  );
    
    // Copy CSR matrix to device
    cudaCheck( hipMemcpy(cu_row_ptr, row_ptr, sizeof(int)    * nrows1,    hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_col,     col,     sizeof(int)    * nnz,       hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_val,     val,     sizeof(double) * nnz,       hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(cu_x,       x,       sizeof(double) * ncols,     hipMemcpyHostToDevice) );
    
    // Initialize cuSPARSE
    hipsparseHandle_t   cu_SpMV_handle = 0;
    hipStream_t       cu_SpMV_stream = 0;
    hipsparseMatDescr_t cu_SpMV_descr  = 0;
    cudaSparseCheck( hipsparseCreate(&cu_SpMV_handle)                                  );
    cudaSparseCheck( hipsparseCreateMatDescr(&cu_SpMV_descr)                           );
    cudaSparseCheck( hipsparseSetStream(cu_SpMV_handle, cu_SpMV_stream)                );
    cudaSparseCheck( hipsparseSetMatType(cu_SpMV_descr, HIPSPARSE_MATRIX_TYPE_GENERAL)  );
    cudaSparseCheck( hipsparseSetMatIndexBase(cu_SpMV_descr, HIPSPARSE_INDEX_BASE_ZERO) );
    
    // Run SpMV on GPU
    hipEvent_t st, et;
    float SpMV_ms;
    hipEventCreate(&st);
    hipEventCreate(&et);
    hipEventRecord(st, cu_SpMV_stream);
    for (int i = 0; i < ntest; i++)
    {
        cudaSparseCheck(hipsparseDcsrmv(
            cu_SpMV_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            nrows, ncols, nnz, &alpha, cu_SpMV_descr, cu_val, 
            cu_row_ptr, cu_col, cu_x, &beta, cu_y
        ));
    }
    hipEventRecord(et, cu_SpMV_stream);
    hipEventSynchronize(et);
    hipEventElapsedTime(&SpMV_ms, st, et);
    
    double GFlops, ut;
    GFlops = 2.0 * (double) nnz / 1000000000.0;
    ut = (double) SpMV_ms / (double) (ntest * 1000);
    printf("cuSPARSE  GPU CSR SpMV done, used time = %lf (ms), %lf GFlops\n", ut * 1000.0, GFlops / ut);
    
    // Copy results to host
    cudaCheck( hipMemcpy(y, cu_y, sizeof(double) * nrows, hipMemcpyDeviceToHost) );
    
    // Free CUDA resources
    cudaCheck( hipFree(cu_row_ptr) );
    cudaCheck( hipFree(cu_col)     );
    cudaCheck( hipFree(cu_val)     );
    cudaCheck( hipFree(cu_x)       );
    cudaCheck( hipFree(cu_y)       );
    cudaSparseCheck( hipsparseDestroy(cu_SpMV_handle)        );
    cudaSparseCheck( hipsparseDestroyMatDescr(cu_SpMV_descr) );
}
