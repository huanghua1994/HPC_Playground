#include <cstdio>
#include <cstring>
#include <cstdlib>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus)                                                      \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
    }

int main(int argc, char **argv)
{
    int local_rank = atoi(getenv("MPI_LOCALRANKID"));
    CUDA_RT_CALL( hipSetDevice(local_rank) );

    MPI_Init(&argc, &argv);

    int my_rank, n_proc, prev_rank, next_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &n_proc);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);
    prev_rank = (my_rank > 0) ? (my_rank - 1) : (n_proc - 1);
    next_rank = (my_rank + 1) % n_proc;

    int arr_size = 1024;
    if (argc >= 2) arr_size = atoi(argv[1]);
    if (arr_size < 0) arr_size = 1024;
    if (my_rank == 0) printf("arr_size = %d\n", arr_size);

    int *dev_arr;
    size_t arr_bytes = sizeof(int) * arr_size;
    CUDA_RT_CALL( hipMalloc(&dev_arr, 4 * arr_bytes) );
    MPI_Barrier(MPI_COMM_WORLD);

    int *prev_recv_ptr = dev_arr + arr_size * 0;
    int *prev_send_ptr = dev_arr + arr_size * 1;
    int *next_send_ptr = dev_arr + arr_size * 2;
    int *next_recv_ptr = dev_arr + arr_size * 3;
    MPI_Request sreq0, sreq1;
    MPI_Isend(prev_send_ptr, arr_size, MPI_INT, prev_rank, 0, MPI_COMM_WORLD, &sreq0);
    MPI_Isend(next_send_ptr, arr_size, MPI_INT, next_rank, 1, MPI_COMM_WORLD, &sreq1);
    MPI_Recv (prev_recv_ptr, arr_size, MPI_INT, prev_rank, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    MPI_Recv (next_recv_ptr, arr_size, MPI_INT, next_rank, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    MPI_Wait(&sreq0, MPI_STATUS_IGNORE);
    MPI_Wait(&sreq1, MPI_STATUS_IGNORE);
    printf("Rank %d finished 1D halo exchange\n", my_rank);
    MPI_Barrier(MPI_COMM_WORLD);

    CUDA_RT_CALL( hipFree(dev_arr) );

    MPI_Finalize();
    return 0;
}