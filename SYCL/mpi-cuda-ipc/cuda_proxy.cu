#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <assert.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_proxy.h"

#define CUDA_RUNTIME_CHECK(statement)                                               \
    do                                                                              \
    {                                                                               \
        hipError_t result = (statement);                                           \
        if (hipSuccess != result)                                                  \
        {                                                                           \
            fprintf(stderr, "[%s:%d] CUDA failed with %s \n", __FILE__, __LINE__,   \
                    hipGetErrorString(result));                                    \
            exit(-1);                                                               \
        }                                                                           \
        assert(hipSuccess == result);                                              \
    } while (0)


int  cuda_get_device()
{
    int device;
    CUDA_RUNTIME_CHECK( hipGetDevice(&device) );
    return device;
}

int  cuda_check_dev_p2p(const int self_dev_id, const int peer_dev_id)
{
    int can_p2p = 0;
    CUDA_RUNTIME_CHECK( hipDeviceCanAccessPeer(&can_p2p, self_dev_id, peer_dev_id) );
    return can_p2p;
}

void cuda_get_ipc_mem_handle(void *dptr, int *handle_bytes, void **handle_)
{
    *handle_bytes = sizeof(hipIpcMemHandle_t);
    hipIpcMemHandle_t *handle_p = (hipIpcMemHandle_t *) malloc(sizeof(hipIpcMemHandle_t));
    CUDA_RUNTIME_CHECK( hipIpcGetMemHandle(handle_p, dptr) );
    *handle_ = handle_p;
}

void cuda_open_ipc_mem_handle(void **dptr, void *handle)
{
    CUDA_RUNTIME_CHECK( hipIpcOpenMemHandle(dptr, *((hipIpcMemHandle_t *) handle), hipIpcMemLazyEnablePeerAccess) );
}

void cuda_close_ipc_mem_handle(void *dptr)
{
    CUDA_RUNTIME_CHECK( hipIpcCloseMemHandle(dptr) );
}